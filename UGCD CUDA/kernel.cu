#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Stdafx.h"
#include "UniformGrid.h"
#include "ObjectLoader.h"
//#include "glut.h"

#define SCREEN_X 100 //Max x size of the grid
#define SCREEN_Y 100 //Max y size of the grid
#define SCREEN_Z 100 //Max z size of the grid

float cameraX = SCREEN_X / 2;
float cameraY = SCREEN_Y / 2;
float cameraZ = SCREEN_Z / 2;
float totalAngleX = 0.0;
float totalAngleY = 0.0;

int col = 0;
bool red = true, green = true, gridShown = true; //Used to toggle the red object, the green objects, and the cell on or off

ObjectLoader obj;
UniformGrid grids;

grid*** cell;

int* x;
int* y;
int* z;

int setgCO = 0;
int collisions = 0;

hipError_t colDetCuda(grid* outputCel, int cellX, int cellY, int cellZ, int* coordX, int* dev_coordY, int* coordZ, unsigned int size, int pass);

__device__ int collisionChecker(grid* dev_cell, int x, int y, int z, int x2, int y2, int z2, int dev_cellX, int dev_cellY, int dev_cellZ)
{
	int collisionCount = 0; //Temporary count for the number of collisions found in the cell

	float distance = 0; //Used to store the distance between the objects being checked
	float comRadius = 0; //Used to store the combined radius of the two objects and then gets squared

	//Loop for the number of object in the cell
	for (int i = 0; i < dev_cell[(x * dev_cellX + y) * dev_cellZ + z].objCount; i++)
	{
		//Loop for the number of objects in the adjacent cell that hasn't been checked yet
		for (int j = 0; j < dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].objCount; j++)
		{
			//Calculate the distance between the two objects using pythagoras without the square root to work in square space
			distance = (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].x - dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].x) * (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].x - dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].x)
				+ (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].y - dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].y) * (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].y - dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].y)
				+ (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].z - dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].z) * (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].z - dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].z);

			//Add the radiuses of the two objects together
			comRadius = dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].r + dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].r;

			//Square the radius to work in square space
			comRadius *= comRadius;

			//Check if distance is less than or equal to the calculated radius - means a collision has occured
			if (distance <= comRadius)
			{
				collisionCount++; //increment the collision counter by 1
				dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].collided = true; //Set the bool the tells the object it's collided to true
				dev_cell[(x2 * dev_cellX + y2) * dev_cellZ + z2].object[j].collided = true; //Set the bool the tells the object it's collided to true
			}
		}
	}

	if (collisionCount > 0)
		return collisionCount; //Return the collision count found in this cell
	else
		return 0;
}

__device__ int currentCellCollisionDetection(grid* dev_cell, int x, int y, int z, int dev_cellX, int dev_cellY, int dev_cellZ)
{
	int collisionCount = 0; //Temporary count for the number of collisions found in the cell

	float distance = 0; //Used to store the distance between the objects being checked
	float comRadius = 0; //Used to store the combined radius of the two objects and then gets squared

	//Loop for the number of object in the cell
	if (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].objCount > 1)
	{
		//Loop for the number of object in the cell
		for (int i = 0; i < dev_cell[(x * dev_cellX + y) * dev_cellZ + z].objCount; i++)
		{
			//Loop for the number of objects in the cell that haven't been checked yet
			for (int j = i + 1; j < dev_cell[(x * dev_cellX + y) * dev_cellZ + z].objCount; j++)
			{
				//Calculate the distance between the two objects using pythagoras without the square root to work in square space
				distance = (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].x - dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].x) * (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].x - dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].x)
					+ (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].y - dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].y) * (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].y - dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].y)
					+ (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].z - dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].z) * (dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].z - dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].z);

				//Add the radiuses of the two objects together
				comRadius = dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].r + dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].r;

				//Square the radius to work in square space
				comRadius *= comRadius;

				//Check if distance is less than or equal to the calculated radius - means a collision has occured
				if (distance <= comRadius)
				{
					collisionCount++; //increment the collision counter by 1
					dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[i].collided = true; //Set the bool the tells the object it's collided to true
					dev_cell[(x * dev_cellX + y) * dev_cellZ + z].object[j].collided = true; //Set the bool the tells the object it's collided to true
				}
			}
		}
	}

	if (collisionCount > 0)
		return collisionCount; //Return the collision count found in this cell
	else
		return 0;
}

__global__ void colDetKernel(grid* dev_cell, gridsConObj* dev_gCO, int dev_cellX, int dev_cellY, int dev_cellZ, int* dev_coordX, int* dev_coordY, int* dev_coordZ, int dev_collision, unsigned int size)
{
	int i = threadIdx.x;

	//Stores if collision detections are needed
	bool ful = false, fum = false, fur = false, fml = false, fmm = false, fmr = false, fdl = false, fdm = false, fdr = false;
	bool mul = false, mum = false, mur = false, mml = false, mmr = false, mdl = false, mdm = false, mdr = false;
	bool bul = false, bum = false, bur = false, bml = false, bmm = false, bmr = false, bdl = false, bdm = false, bdr = false;

	//Check if collision detection is needed for the 9 cells infront of the current cell
	//Check if the cell being looked at isn't at the very front of the grid
	if (dev_gCO[dev_coordZ[i]].z > 0)
	{
		//Check if the cell being looked at is below the top of the grid
		if (dev_gCO[dev_coordY[i]].y < dev_cellY - 1)
		{
			//Check if cell being looked at isn't on the far left of the grid
			if (dev_gCO[dev_coordX[i]].x > 0)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
					ful = true;
			}

			//Check if a collision detection is needed
			if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
				fum = true;

			//Check if cell being looked at isn't on the far right of the grid
			if (dev_gCO[dev_coordX[i]].x < dev_cellX - 1)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
					fur = true;
			}
		}

		//Check if cell being looked at isn't on the far left of the grid
		if (dev_gCO[dev_coordX[i]].x > 0)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
				fml = true;
		}

		//Check if a collision detection is needed
		if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
			fmm = true;

		//Check if cell being looked at isn't on the far right of the grid
		if (dev_gCO[dev_coordX[i]].x < dev_cellX - 1)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
				fmr = true;
		}

		//Check if the cell being looked at is above the bottom of the grid
		if (dev_gCO[dev_coordY[i]].y > 0)
		{
			//Check if cell being looked at isn't on the far left of the grid
			if (dev_gCO[dev_coordX[i]].x > 0)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
					fdl = true;
			}
			if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
				fdm = true;

			//Check if cell being looked at isn't on the far right of the grid
			if (dev_gCO[dev_coordX[i]].x < dev_cellX - 1)
			{
				//+1 -1 -1
				if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z - 1))].objCount > 0)
					fdr = true;
			}
		}
	}
	//Check if collision detection is needed for the 9 cells infront of the current cell

	//Check if collision detection is needed for the 8 cells around the current cell
	//Check if the cell being looked at is below the top of the grid
	if (dev_gCO[dev_coordY[i]].y < dev_cellY - 1)
	{
		//Check if cell being looked at isn't on the far left of the grid
		if (dev_gCO[dev_coordX[i]].x > 0)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
				mul = true;
		}

		//Check if a collision detection is needed
		if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
			mum = true;

		//Check if cell being looked at isn't on the far right of the grid
		if (dev_gCO[dev_coordX[i]].x < dev_cellX - 1)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
				mur = true;
		}
	}

	//Check if cell being looked at isn't on the far left of the grid
	if (dev_gCO[dev_coordX[i]].x > 0)
	{
		if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
			mml = true;
	}

	//Check if cell being looked at isn't on the far right of the grid
	if (dev_gCO[dev_coordX[i]].x < dev_cellX - 1)
	{
		if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
			mmr = true;
	}

	//Check if the cell being looked at is above the bottom of the grid
	if (dev_gCO[dev_coordY[i]].y > 0)
	{
		//Check if cell being looked at isn't on the far left of the grid
		if (dev_gCO[dev_coordX[i]].x > 0)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
				mdl = true;
		}
		if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
			mdm = true;

		//Check if cell being looked at isn't on the far right of the grid
		if (dev_gCO[dev_coordX[i]].x < dev_cellX - 1)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].objCount > 0)
				mdr = true;
		}
	}
	//Check if collision detection is needed for the 8 cells around the current cell

	//Check if collision detection is needed for the 9 cells behind the current cell
	//Check if the cell being looked at isn't at the very back of the grid
	if (dev_gCO[dev_coordZ[i]].z != dev_cellZ - 1)
	{
		//Check if the cell being looked at is below the top of the grid
		if (dev_gCO[dev_coordY[i]].y != dev_cellY - 1)
		{
			//Check if cell being looked at isn't on the far left of the grid
			if (dev_gCO[dev_coordX[i]].x > 0)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
					bul = true;
			}
			if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
				bum = true;

			//Check if cell being looked at isn't on the far right of the grid
			if (dev_gCO[dev_coordX[i]].x != dev_cellX - 1)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y + 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
					bur = true;
			}
		}

		//Check if cell being looked at isn't on the far left of the grid
		if (dev_gCO[dev_coordX[i]].x > 0)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
				bml = true;
		}

		if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
			bmm = true;

		//Check if cell being looked at isn't on the far right of the grid
		if (dev_gCO[dev_coordX[i]].x != dev_cellX - 1)
		{
			if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
				bmr = true;
		}

		//Check if the cell being looked at is above the bottom of the grid
		if (dev_gCO[dev_coordY[i]].y != 0)
		{
			//Check if cell being looked at isn't on the far left of the grid
			if (dev_gCO[dev_coordX[i]].x != 0)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x - 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
					bdl = true;
			}
			if (dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
				bdm = true;

			//Check if cell being looked at isn't on the far right of the grid
			if (dev_gCO[dev_coordX[i]].x != dev_cellX - 1)
			{
				if (dev_cell[((dev_gCO[dev_coordX[i]].x + 1) * dev_cellX + ((dev_gCO[dev_coordY[i]].y - 1)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z + 1))].objCount > 0)
					bdr = true;
			}
		}

	}
	//Check if collision detection is needed for the 9 cells behind the current cell

	//Actual Collision Detection
	//Front 9 cells
	if (ful)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	if (fum)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	if (fur)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);

	if (fml)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	if (fmm)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	if (fmr)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);

	if (fdl)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	if (fdm)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	if (fdr)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z - 1, dev_cellX, dev_cellY, dev_cellZ);
	//Front 9 cells

	//Middle 8 cells
	if (mul)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);
	if (mum)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);
	if (mur)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);

	if (mml)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);
	if (mmr)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);

	if (mdl)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);
	if (mdm)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);
	if (mdr)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);
	//Middle 8 cells

	//Back 9 cells
	if (bul)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	if (bum)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	if (bur)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y + 1, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);

	if (bml)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	if (bmm)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	if (bmr)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);

	if (bdl)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x - 1, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	if (bdm)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	if (bdr)
		dev_collision += collisionChecker(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_gCO[dev_coordX[i]].x + 1, dev_gCO[dev_coordY[i]].y - 1, dev_gCO[dev_coordZ[i]].z + 1, dev_cellX, dev_cellY, dev_cellZ);
	//Back 9 cells

	dev_collision += currentCellCollisionDetection(dev_cell, dev_gCO[dev_coordX[i]].x, dev_gCO[dev_coordY[i]].y, dev_gCO[dev_coordZ[i]].z, dev_cellX, dev_cellY, dev_cellZ);

	dev_cell[((dev_gCO[dev_coordX[i]].x) * dev_cellX + ((dev_gCO[dev_coordY[i]].y)) * dev_cellZ + (dev_gCO[dev_coordZ[i]].z))].collisions = dev_collision;
}

void init()
{
	int objCount; //Stores the number of objects read from a file

	objCount = obj.objectCounter(); //Counts the number of objects to be loaded in

	grids.setobj(objCount); //Reads in the objects from the file

	float rad; //Stores the radius of the largest object

	rad = obj.objLoader(grids);

	grids.objCounter /= 4; //Divide the object counter by 4 since each object has four variables read in, making the count 4 times larger than it should be

	grids.setGrid(SCREEN_X, SCREEN_Y, SCREEN_Z, rad); //Creates the grid and cells

	grids.setObjectsInGrid(); //Add the objects to the cells they belong to

	x = new int[grids.objCounter]; //Set the size of the x array
	y = new int[grids.objCounter]; //Set the size of the y array
	z = new int[grids.objCounter]; //Set the size of the z array

	cell = grids.getGrid();

	int itemCount = 0;

	//Loop for the number of cells in the X
	for (int i = 0; i < grids.getCellNumX(); i++)
	{
		//Loop for the number of cells in the Y
		for (int j = 0; j < grids.getCellNumY(); j++)
		{
			//Loop for the number of cells in the Z
			for (int k = 0; k < grids.getCellNumZ(); k++)
			{
				//Check if the object count for the cell being looked at isn't 0
				if (cell[i][j][k].objCount != 0)
				{
					//Set the corresponding gCO x, y, z values to i, j, and k
					grids.gCO[setgCO].x = i;
					grids.gCO[setgCO].y = j;
					grids.gCO[setgCO].z = k;

					setgCO++; //Increment setgCO by 1

							  //Loop for the number of objects in the cell being looked at
					for (int l = 0; l < cell[i][j][k].objCount; l++)
					{
						x[itemCount] = i;
						y[itemCount] = j;
						z[itemCount] = k;

						itemCount++; //Increment itemCount by 1
					}
				}
			}
		}
	}

	//Output initialisation values - DEBUGGING
	cout << "CELL X: " << grids.getCellNumX() << endl;
	cout << "CELL Y: " << grids.getCellNumY() << endl;
	cout << "CELL Z: " << grids.getCellNumZ() << endl;
	cout << "SIZE OF GRID: " << grids.getCellNumX() * grids.getCellNumY() * grids.getCellNumZ() << endl;
	cout << "OBJECTS IN CELLS: " << grids.objCounter << endl;
	cout << "OBJECTS SAVED: " << itemCount << endl;
	cout << "CELLS CONTAINING OBJECTS: " << setgCO << endl;
	cout << "FINISHED INITIALISING" << endl;

	//collisionDetection();

	//glutMain(argc, argv);
}

int main()
{
	grid* outputCel; //Used to store the grid to be output from the kernel

	init(); //Handles initialising the objects and grid

	//Handles storing the number of threads to be used in each pass
	int passArray[27] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

	//Loop for the number of cells in the x
	for (int i = 0; i < grids.cellNumX; i++)
	{
		//Loop for the number of cells in the y
		for (int j = 0; j < grids.cellNumY; j++)
		{
			//Loop for the number of cells in the z
			for (int k = 0; k < grids.cellNumZ; k++)
			{
				//Increment the pass being looked at's counter by 1
				passArray[cell[i][j][k].pass - 1]++;
			}
		}
	}

	//Dynamically allocate the size of the outputCel
	outputCel = new grid[grids.getCellNumX() * grids.getCellNumY() * grids.getCellNumZ()];

	//Loop for the number of passes to be done
	for (int l = 1; l < 28; l++)
	{
		int* xCoords; //Used to store the x position of the cells for whatever pass being looked at
		int* yCoords; //Used to store the y position of the cells for whatever pass being looked at
		int* zCoords; //Used to store the z position of the cells for whatever pass being looked at

		//Dynamically allocate the size of xCoords, yCoords, and zCoords
		xCoords = new int[passArray[l - 1]];
		yCoords = new int[passArray[l - 1]];
		zCoords = new int[passArray[l - 1]];

		int coordCount = 0;

		//Loop for the number of cells in the x
		for (int i = 0; i < grids.cellNumX; i++)
		{
			//Loop for the number of cells in the y
			for (int j = 0; j < grids.cellNumY; j++)
			{
				//Loop for the number of cells in the z
				for (int k = 0; k < grids.cellNumZ; k++)
				{
					//Check if the pass of the cell being looked at matches the pass being looked at
					if (cell[i][j][k].pass == l)
					{
						//Set the coordinates of the cell
						xCoords[coordCount] = i;
						yCoords[coordCount] = j;
						zCoords[coordCount] = k;

						coordCount++; //Increment the coordCount by 1
					}
				}
			}
		}

		//Call the colDetCuda function which handles setting up the kernels to be used
		hipError_t cudaStatus = colDetCuda(outputCel, grids.getCellNumX(), grids.getCellNumY(), grids.getCellNumZ(), xCoords, yCoords, zCoords, passArray[l - 1], l);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		
		//Loop for the number of cells in the x
		for (int i = 0; i < grids.getCellNumX(); i++)
		{
			//Loop for the number of cells in the y
			for (int j = 0; j < grids.getCellNumY(); j++)
			{
				//Loop for the number of cell in the z
				for (int k = 0; k < grids.getCellNumZ(); k++)
				{
					//Check if the pass value in the outputCel is the same as the current pass
					if (outputCel[(i * grids.getCellNumX() + j) * grids.getCellNumZ() + k].pass = l)
					{
						//Add the number of collisions done to the total number of collisions
						collisions += outputCel[(i * grids.getCellNumX() + j) * grids.getCellNumZ() + k].collisions;
					}
				}
			}
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
	}

	//cout << "Test Value: " << cellC << endl;
	cout << "Collisions: " << collisions << endl;
    return 0;
}

// Helper function for using CUDA to perform uniform grid collision detection in parallel
hipError_t colDetCuda(grid* outputCel, int cellX, int cellY, int cellZ, int* coordX, int* coordY, int* coordZ, unsigned int size, int pass)
{
    int *dev_coordX = 0; //coordX to be used on the device
    int *dev_coordY = 0; //coordY to be used on the device
    int *dev_coordZ = 0; //coordZ to be used on the device
	int dev_cellX = cellX; //cellX to be used on the device
	int dev_cellY = cellY; //cellY to be used on the device
	int dev_cellZ = cellZ; //cellZ to be used on the device
	int dev_collision = 0; //collision to be used on the device

	gridsConObj* dev_gCO; //gCO to be used on the device
	grid* cel; //cel to be used on the device
	grid* dev_cell; //cell to be used on the device

    hipError_t cudaStatus;

	//Dynamically allocate the size of the cel
	cel = new grid[cellX * cellY * cellZ];

	//Loop for the number of cells in the x
	for (int i = 0; i < cellX; i++)
	{
		//Loop for the number of cells in the y
		for (int j = 0; j < cellY; j++)
		{
			//Loop for the number of cells in the z
			for (int k = 0; k < cellZ; k++)
			{
				//Copy the values in cell to cel
				cel[(i * cellX + j) * cellZ + k] = cell[i][j][k];
			}
		}
	}

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_cell, size * sizeof(grid));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//Loop for the number of cells in the x
	for (int i = 0; i < cellX; i++)
	{
		//Loop for the number of cells in the y
		for (int j = 0; j < cellY; j++)
		{
			//Loop for the number of cells in the z
			for (int k = 0; k < cellZ; k++)
			{
				cudaStatus = hipMalloc((void**)&cel[(i * cellX + j) * cellZ + k].object, size * sizeof(objects));
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMalloc failed!");
					goto Error;
				}

				cudaStatus = hipMemcpy(cel[(i * cellX + j) * cellZ + k].object, cell[i][j][k].object, size * sizeof(objects), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
					goto Error;
				}
			}
		}
	}

	cudaStatus = hipMalloc((void**)&dev_gCO, size * sizeof(gridsConObj));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    cudaStatus = hipMalloc((void**)&dev_coordX, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_coordY, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_coordZ, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_cell, cel, size * sizeof(grid), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(&dev_cell->object, &cel->object, size * sizeof(objects), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_gCO, grids.gCO, size * sizeof(gridsConObj), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_coordX, coordX, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_coordY, coordY, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_coordZ, coordZ, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
    colDetKernel<<<1, size>>>(dev_cell, dev_gCO, dev_cellX, dev_cellY, dev_cellZ, dev_coordX, dev_coordY, dev_coordZ, dev_collision, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(&outputCel->object, &dev_cell->object, size * sizeof(objects), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(outputCel, dev_cell, size * sizeof(grid), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
Error:
	hipFree(dev_coordX);
	hipFree(dev_coordY);
	hipFree(dev_coordZ);

	hipFree(dev_cell);
	hipFree(dev_gCO);
	hipFree(cel->object);
    return cudaStatus;
}